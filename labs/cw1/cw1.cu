#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <cstring>  // For strcmp (used on the host side)
#include <algorithm> // For std::transform (used on the host side)


using namespace std;

// A device function to compare two strings for equality
__device__ bool gpu_strncmp(const char *str1, const char *str2, int n) {
    for (int i = 0; i < n; ++i) {
        if (str1[i] != str2[i]) {
            return false;
        }
    }
    return true;
}

__global__ void calc_token_occurrences_kernel(char *data, int dataSize, char *token, int tokenLen, int *numOccurrences)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dataSize - tokenLen) {
        // Use custom GPU function to compare token with the substring in data
        if (gpu_strncmp(&data[idx], token, tokenLen)) {
            // Check if the match is a standalone word
            bool validPrefix = (idx == 0) || (data[idx - 1] < 'a' || data[idx - 1] > 'z');
            bool validSuffix = (idx + tokenLen >= dataSize) || (data[idx + tokenLen] < 'a' || data[idx + tokenLen] > 'z');
            if (validPrefix && validSuffix) {
                atomicAdd(numOccurrences, 1);
            }
        }
    }
}

int main(int argc, char **argv)
{
    //chosen file
    const char *filepath = "C:/set10108-cw/set10108/labs/cw1/dataset/shakespeare.txt";
    ifstream file(filepath, ios::binary);

    if (!file) {
        cerr << "Error: Could not open the file " << filepath << endl;
        return -1;
    }

    // Read file into buffer
    file.seekg(0, ios::end);
    streamsize fileSize = file.tellg();
    file.seekg(0, ios::beg);
    char* fileData = new char[fileSize]; // Allocate a raw char array
    if (!file.read(fileData, fileSize)) {
        cerr << "Error: Could not read the file content." << endl;
        delete[] fileData; // Free allocated memory
        return -1;
    }

    // Convert to lowercase using a loop instead of std::transform
    for (int i = 0; i < fileSize; ++i) {
        fileData[i] = tolower(fileData[i]);
    }

    // Define an array of tokens to search for
    const char* tokens[] = {"sword", "fire", "death", "love", "hate", "the", "man", "woman"};
    int numTokens = sizeof(tokens) / sizeof(tokens[0]);

    // Add header for GPU results in the console
    cout << "GPU Results:" << endl;  // Console output to indicate GPU results

    // Allocate memory on GPU for the data buffer
    char *d_data;
    hipError_t err = hipMalloc((void**)&d_data, fileSize);
    if (err != hipSuccess) {
        cerr << "Error allocating device memory for data: " << hipGetErrorString(err) << endl;
        delete[] fileData;
        return -1;
    }
    hipMemcpy(d_data, fileData, fileSize, hipMemcpyHostToDevice);

       // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Loop through each token
    for (int i = 0; i < numTokens; ++i) {
        const char *token = tokens[i];
        int tokenLen = strlen(token);

        // Allocate memory on GPU for the token
        char *d_token;
        err = hipMalloc((void**)&d_token, tokenLen);
        if (err != hipSuccess) {
            cerr << "Error allocating device memory for token '" << token << "': " << hipGetErrorString(err) << endl;
            continue; // Skip this word if allocation fails
        }
        hipMemcpy(d_token, token, tokenLen, hipMemcpyHostToDevice);

        // Allocate memory for the occurrences count
        int numOccurrences = 0;
        int *d_numOccurrences;
        err = hipMalloc((void**)&d_numOccurrences, sizeof(int));
        if (err != hipSuccess) {
            cerr << "Error allocating device memory for numOccurrences for token '" << token << "': " << hipGetErrorString(err) << endl;
            hipFree(d_token);
            continue; // Skip this word if allocation fails
        }
        hipMemcpy(d_numOccurrences, &numOccurrences, sizeof(int), hipMemcpyHostToDevice);

        // Launch kernel
        int blockSize = 256;
        int gridSize = (fileSize + blockSize - 1) / blockSize;
        calc_token_occurrences_kernel<<<gridSize, blockSize>>>(d_data, fileSize, d_token, tokenLen, d_numOccurrences);

        // Record stop event and calculate elapsed time
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        // Synchronize and check for errors after kernel launch
        hipError_t errSync = hipDeviceSynchronize();
        if (errSync != hipSuccess) {
            cerr << "Error during kernel execution for token '" << token << "': " << hipGetErrorString(errSync) << endl;
            hipFree(d_token);
            hipFree(d_numOccurrences);
            continue; // Skip this word if an error occurs
        }

        // Copy result back to host
        hipMemcpy(&numOccurrences, d_numOccurrences, sizeof(int), hipMemcpyDeviceToHost);

       // Print debug statement to ensure token has been processed
        cout << "Processed token '" << token << "' with occurrences: " << numOccurrences << " in " << milliseconds << " ms" << endl;

        // Free memory allocated for token and occurrences count
        hipFree(d_token);
        hipFree(d_numOccurrences);
    }

    // Free memory
    hipFree(d_data);
    delete[] fileData; // Free allocated memory

    return 0;
}
